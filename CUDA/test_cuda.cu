#include "hip/hip_runtime.h"
#include <stdio.h>

#include "api.h"

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
extern "C" int addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

extern "C" int resetCudaDevice() {
    hipError_t cudaStatus;
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

// __global__ void Traverse(uchar3 *_src_dev, uchar3 *dst_dev, int col, int row) {
//     // 一维数据索引计算（万能计算方法）
//     int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) +
//               blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) +
//               blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.z * (blockDim.x * blockDim.y) +
//               threadIdx.y * blockDim.x + threadIdx.x;
//     if (tid < col * row * 3) {
//         dst_dev[tid].x = 0.3 * _src_dev[tid].x;
//         dst_dev[tid].y = 0.6 * _src_dev[tid].y;
//         dst_dev[tid].z = 0.1 * _src_dev[tid].z;
//     }
// }

// __global__ void Traverse(uchar3 *_src_dev, uchar3 *dst_dev, int col, int row) {
//     // 一维数据索引计算（万能计算方法）
//     int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) +
//               blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) +
//               blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.z * (blockDim.x * blockDim.y) +
//               threadIdx.y * blockDim.x + threadIdx.x;
//     if (tid < col * row * 3) {
//         dst_dev[tid].x = 255 - _src_dev[tid].x;
//         dst_dev[tid].y = 255 - _src_dev[tid].y;
//         dst_dev[tid].z = 255 - _src_dev[tid].z;
//     }
// }

__global__ void Traverse(uchar3 *_src_dev, uchar3 *dst_dev, int col, int row) {
    // 一维数据索引计算（万能计算方法）
    int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) +
              blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) +
              blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.z * (blockDim.x * blockDim.y) +
              threadIdx.y * blockDim.x + threadIdx.x;
    if (tid < col * row * 3) {
        dst_dev[tid].x = 0.3 * _src_dev[tid].x;
        dst_dev[tid].y = 0.6 * _src_dev[tid].y;
        dst_dev[tid].z = 0.1 * _src_dev[tid].z;
    }
}

extern "C" int processWithCuda(uint8_t *src_data, uint32_t rows, uint32_t cols, uint8_t *dst_data) {
    uchar3 *src_dev, *dst_dev;
    hipMalloc((void **)&src_dev, rows * cols * sizeof(uchar3));
    hipMalloc((void **)&dst_dev, rows * cols * sizeof(uchar3));
    hipMemcpy(src_dev, src_data, rows * cols * sizeof(uchar3), hipMemcpyHostToDevice);
    dim3 grid(1 + (cols * rows / (32 * 32 + 1)), 1, 1); // grid
    dim3 block(32, 32, 1);

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1, 0);
    Traverse<<<grid, block>>>(src_dev, dst_dev, cols, rows);
    hipEventSynchronize(stop1);
    hipEventRecord(stop1, 0);
    float time1;
    hipEventElapsedTime(&time1, start1, stop1);
    printf("Gpu所耗费的时间: %fms\n", time1);
    hipMemcpy(dst_data, dst_dev, rows * cols * sizeof(uchar3), hipMemcpyDeviceToHost);

    // free
    hipFree(src_dev);
    hipFree(dst_dev);
    return 0;
}

__global__ void rgb2grayincuda(uchar3 *const d_in, uint8_t *const d_out, uint rows, uint cols) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) {
        uchar3 rgb = d_in[idy * cols + idx];
        d_out[idy * cols + idx] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
    }
}

extern "C" void grayWithCuda(uint8_t *src_data, uint32_t rows, uint32_t cols, uint8_t *dst_data) {
    uchar3 *d_in;
    unsigned char *d_out;

    hipMalloc((void **)&d_in, rows * cols * sizeof(uchar3));
    hipMalloc((void **)&d_out, rows * cols * sizeof(uint8_t));

    hipMemcpy(d_in, src_data, rows * cols * sizeof(uchar3), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32, 1);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    clock_t start, end;
    start = clock();

    rgb2grayincuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, rows, cols);

    hipDeviceSynchronize();
    end = clock();

    printf("cuda exec time is %.8f\n", (double)(end - start) / CLOCKS_PER_SEC);

    hipMemcpy(dst_data, d_out, rows * cols * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}
